

#include <hip/hip_runtime.h>
#include <iostream>

// Simple CUDA kernel that adds two arrays


__global__ void addKernel(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
      c[idx] = a[idx] + b[idx];
    }
  }
  
  int main() {
    int n = 1024;
    float *a, *b, *c;
    float *d_a, *d_b, *d_c;
  
    // Allocate host memory
    a = (float *)malloc(n * sizeof(float));
    b = (float *)malloc(n * sizeof(float));
    c = (float *)malloc(n * sizeof(float));
  
    // Initialize host memory
    for (int i = 0; i < n; i++) {
      a[i] = i;
      b[i] = i * 2;
    }
  
    // Allocate device memory
    hipMalloc((void **)&d_a, n * sizeof(float));
    hipMalloc((void **)&d_b, n * sizeof(float));
    hipMalloc((void **)&d_c, n * sizeof(float));
  
    // Copy host memory to device memory
    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);
  
    // Launch kernel
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    addKernel<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);
  
    // Copy device memory back to host memory
    hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);
  
    // Print results
    for (int i = 0; i < n; i++) {
      std::cout << a[i] << " + " << b[i] << " = " << c[i] << std::endl;
    }
  
    // Free memory
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
  
    return 0;
  }
